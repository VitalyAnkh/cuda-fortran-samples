// Filename: index.cu
// nvcc -c -arch sm_35 key.cu
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/sort.h>
#include <hip/hip_runtime_api.h>

using namespace std;

__device__ int *a_i;
__device__ float *a_f;
__device__ double *a_d;

struct cmpi : public binary_function<int, int, bool>
{
  __device__  bool operator()(const int i, const int j) const
  {return ( a_i[i] < a_i[j]);}
};

struct cmpf : public binary_function<int, int, bool>
{
  __device__  bool operator()(const int i, const int j) const
  {return ( a_f[i] < a_f[j]);}
};

struct cmpd : public binary_function<int, int, bool>
{
  __device__  bool operator()(const int i, const int j) const
  {return ( a_d[i] < a_d[j]);}
};

// index sort for integer arrays
void thrust_int_sort_wrapper( int *dev_data, int *dev_idx, int N, hipStream_t stream)
{
thrust::device_ptr <int> dev_ptr(dev_idx);
hipMemcpyToSymbolAsync(HIP_SYMBOL(a_i), &dev_data,
                        sizeof(int *), (size_t) 0,
                        hipMemcpyHostToDevice, stream);
thrust::sequence(thrust::cuda::par.on(stream), dev_ptr, dev_ptr+N);
thrust::sort(thrust::cuda::par.on(stream), dev_ptr, dev_ptr+N, cmpi());
}

// index sort for float arrays
void thrust_float_sort_wrapper( float *dev_data, int *dev_idx, int N, hipStream_t stream)
{
thrust::device_ptr <int> dev_ptr(dev_idx);
hipMemcpyToSymbolAsync(HIP_SYMBOL(a_f), &dev_data,
                        sizeof(float *), (size_t) 0,
                        hipMemcpyHostToDevice, stream);
thrust::sequence(thrust::cuda::par.on(stream), dev_ptr, dev_ptr+N);
thrust::sort(thrust::cuda::par.on(stream), dev_ptr, dev_ptr+N, cmpf());
}

// index sort for double arrays
void thrust_double_sort_wrapper( double * dev_data, int *dev_idx, int N, hipStream_t stream)
{
thrust::device_ptr <int> dev_ptr(dev_idx);
hipMemcpyToSymbolAsync(HIP_SYMBOL(a_d), &dev_data,
                        sizeof(double *), (size_t) 0,
                        hipMemcpyHostToDevice, stream);
thrust::sequence(thrust::cuda::par.on(stream), dev_ptr, dev_ptr+N);
thrust::sort(thrust::cuda::par.on(stream), dev_ptr, dev_ptr+N, cmpd());
}
